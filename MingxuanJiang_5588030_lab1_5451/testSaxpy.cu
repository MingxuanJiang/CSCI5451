
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <omp.h>
#include <unistd.h>

/*-------------------- POSIX-compliant timer in seconds */
// calculate the time with function wctime()
double wctime() 
{
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec + 1E-6 * tv.tv_usec);
}

// initialize the length of vector
#define NN 524288

// calculate the form
__global__ void saxpy_par(int N, float a, float *A, float *B){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N){
       B[i] = a * A[i] + B[i]; 
    }
}

float saxpy_check(int n, float a, float *x, float *y, float *z) {
    // a, x, y == original data for saxpy
    // z = result found -- with which to compare.
    float s=0.0, t = 0.0;
    for (int i=0; i<n; i++) {
        y[i] += a * x[i] ;
        s += (y[i] - z[i])*(y[i] - z[i]);
        t += z[i]*z[i];
    }
    if (t == 0.0) return(-1);
    else
    return(sqrt(s/t));
}

int main(){
    float *x_d, *y_d, *z_d;    // vector x, y, z in GPUs
    float *x, *y, *z;          // vector x, y, z in CPUs
    float a = 1, Mflops, err;
    double t1, t2;             // time
    int N, i, vecLen, iter;
    int MatSize;               // the size of vector
    //-------------------- set dimension N
    N = NN;
    MatSize = N*sizeof(float);
    //-------------------- allocate on cpu
    x = (float *)malloc(MatSize);        
    y = (float *)malloc(MatSize);        
    z = (float *)malloc(MatSize);    
    if ((x==NULL) | (y==NULL) | (z==NULL) ) 
        exit(1); 
    //-------------------- allocate on GPU
    if (hipMalloc((void **) &x_d, MatSize) != hipSuccess) 
        exit(2);	      
    if (hipMalloc((void **) &y_d, MatSize) != hipSuccess) 
        exit(3);	      
    if (hipMalloc((void **) &z_d, MatSize) != hipSuccess) 
        exit(4);
    
    //set vector x,y with random numbers
    for(i = 0 ; i < N ; i ++)	{
        x[i]  = (float) rand() / (float) rand();
        y[i]  = (float) rand() / (float) rand();
    }
    int NITER = 100;
    a = a/(float) NITER;
    for(vecLen = 1024; vecLen <= N; vecLen *= 2){
        //-------------------- copy matrices x,y to GPU memory
        hipMemcpy(x_d, x, sizeof(float)*vecLen, hipMemcpyHostToDevice);
        hipMemcpy(y_d, y, sizeof(float)*vecLen, hipMemcpyHostToDevice);
        // set dimension of block and grid
        dim3 dimBlock = dim3(1024);
        dim3 dimGrid = dim3(vecLen/1024);
        t1 = wctime(); // record time here
        for(iter = 0;iter<NITER;iter++){
            saxpy_par<<<dimGrid,dimBlock>>>(vecLen,a,x_d,y_d);            
        }
        t2 = wctime(); // record exit time here
        hipMemcpy(z, y_d, sizeof(float)*vecLen, hipMemcpyDeviceToHost);
        Mflops = 2*vecLen*NITER*1E-6/(t2-t1);
        //check error
        err = 0.0;
        err = saxpy_check(vecLen,1,x,y,z);
        // print results for this vecLen...
        printf("** vecLen = %d, Mflops = %.2f, err = %.2e\n",vecLen,Mflops,err);
    }
    //-------------------- Free Host arrays
    free(x); 
    free(y);
    free(z);
    //-------------------- Free GPU memory
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);	
    return 0;
}




